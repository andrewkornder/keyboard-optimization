#include <lib/def.cuh>
#include <lib/text.cuh>
#include <common.cuh>
#include <iostream>
#include <metric.cuh>
#include <unordered_map>
#include <vector>

class TestGroup {
    std::vector<std::string> names;
    std::vector<stats> scores;

public:
    void combine(const TestGroup &other) {
        for (int i = 0; i < other.names.size(); ++i) {
            names.push_back(other.names[i]);
            scores.push_back(other.scores[i]);
        }
    }

    void add(const std::string &name, const stats score) {
        names.push_back(name);
        scores.push_back(score);
    }

    static void show(const std::string &string, const stats &out) {
        printf("%s:\n", string.c_str());
        printf("    %-13s = %s\n", "score", F3(out.score));
        if constexpr (stats::attrs > 0) {
            const text_t* arr = (text_t*) ((char*) &out + sizeof(out.score));
            for (int i = 0; i < stats::attrs; ++i) {
                printf("    %-13s = %s\n", stats::names[i], F3(arr[i]));
            }
        }
        printf("\n");
    }

    void print() const {
        const int n = scores.size();
        std::vector seen(n, false);

        for (int i = 0; i < n; ++i) {
            int best = -1;
            score_t cmp = 0;
            for (int j = 0; j < n; ++j) {
                if (seen[j]) continue;

                if (best == -1 || CMP(scores[j].score < cmp)) {
                    best = j;
                    cmp = scores[j].score;
                }
            }
            show(names[best], scores[best]);
            seen[best] = true;
        }
    }
};

struct TestCollection {
    std::unordered_map<std::string, std::vector<char>> lut{};

    TestCollection() {
        add(
            "dist-w&p",
            "1234567890"
            "j.glykcub;"
            "ietrhdnosa"
            "/xwmzpf,vq"
        );
        add(
            "dist-full",
            "1234567890"
            "jbglykcu,;"
            "istrhdnoea"
            "/vwmzpf.xq"
        );
        add(
            "okp1",
            "1234567890"
            "/oupqxlbfw"
            "eainkmhtsc"
            ";.,yzjrdvg"
        );
        add(
            "okp2",
            "1234567890"
            ".uofqjmdbg"
            "ieanylrhts"
            ",;/pzwcvkx"
        );
        add(
            "qwerty",
            "1234567890"
            "qwertyuiop"
            "asdfghjkl;"
            "zxcvbnm,./"
        );
        add(
            "alphabet",
            "1234567890"
            "abcdefghij"
            "klmnopqrs;"
            "tuvwxyz,./"
        );
        add(
            "dvorak",
            "1234567890"
            "/,.pyfgcrl"
            "aoeuidhtns"
            ";qjkxbmwvz"
        );
        add(
            "colemak",
            "1234567890"
            "qwfpgjluy;"
            "arstdhneio"
            "zxcvbkm,./"
        );
        add(
            "carpalx",
            "1234567890"
            "qgmlwbyuv;"
            "dstnriaeoh"
            "zxcfjkp,./"
        );
        add(
            "arensito",
            "1234567890"
            "ql,p/;fudk"
            "arenbgsito"
            "zw.hjvcymx"
        );
        add(
            "asset",
            "1234567890"
            "qwjfgypul;"
            "asetdhnior"
            "zxcvbkm,./"
        );
        add(
            "capewell",
            "1234567890"
            ".ywdfjpluq"
            "aersgbtnio"
            "xzcv;kwh,/"
        );
    }

    void add(const std::string &name, const char* array) {
        std::vector<char> &arr = lut[name];
        arr.clear();
        for (const char* ptr = array; *ptr != '\0'; ++ptr) {
            arr.push_back(*ptr);
        }
    }

    void remove(const std::string &name) {
        lut.erase(name);
    }

    void test(TestGroup &tg) const {
        for (const auto &[name, _] : lut) {
            test(tg, name);
        }
    }

    void test(TestGroup &tg, const std::string &name) const {
        if (lut.find(name) == lut.end()) {
            return;
        }

        char t[KEYS];
        const std::vector<char> &arr = lut.at(name);

        int k = 0;
        for (const char c : arr) {
            if (const int j = letterUtils.positionOf(c); j != -1) {
                t[j] = k++;
            }
        }
        if (k != KEYS) {
            printf("Keyboard layout '%s' must have %d valid keys: Found %d.\n", name.c_str(), k, KEYS);
        } else {
            tg.add(name, score(t));
        }
    }
} tests;


void testAll() {
    TestGroup t;
    tests.test(t);
    t.print();
}


bool testable(const std::string &name) {
    return tests.lut.find(name) != tests.lut.end();
}
void test(const std::string &name) {
    TestGroup t;
    tests.test(t, name);
    t.print();
}

void testUser() {
    printf("To test a keyboard, enter a string of text with exactly %d letters included in"
           "the following string:\n%s\n%s\nTo exit the testing loop, enter \"stop\".\n\n" , KEYS, KEYS_LOWER, KEYS_UPPER);
    while (true) {
        char keyboard[KEYS];
        int seen = 0;

        printf("Enter a keyboard or \"stop\"\n");
        while (seen != KEYS) {
            char buffer[1024];
            std::cin.getline(buffer, 1024);
            if (std::string(buffer) == "stop") {
                printf("Exiting.\n");
                goto exit;
            }
            for (int i = 0; buffer[i] != '\0'; ++i) {
                if (letterUtils.positionOf(buffer[i]) != -1) {
                    if (seen < KEYS) keyboard[seen++] = buffer[i];
                    else seen++;
                }
            }
            if (seen > KEYS) {
                printf("Too many keys found. Expected %d, found at least %d.\n", KEYS, seen);
                break;
            }
        }

        if (seen != KEYS) continue;

        printf("Your keyboard:\n");
        for (int i = 0; i < KEYS; ++i) {
            printf("%c", keyboard[i]);
            if (i % 10 == 9) printf("\n");
        }
        printf("\n");

        TestGroup tg;
        tests.add("result", keyboard);
        tests.test(tg, "result");
        tg.print();
    }
    exit:
    tests.remove("result");
}