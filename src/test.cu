#include <lib/def.cuh>
#include <lib/text.cuh>
#include <common.cuh>
#include <iostream>
#include <metric.cuh>
#include <vector>


stats test(const char* kb) {
    char t[KEYS];
    mapKeys(kb, t);

    return score(t);
}

void show(const std::string &string, const stats &out) {
    const text_t* arr = (text_t*) ((char*) &out + sizeof(out.score));

    printf("%s:\n", string.c_str());
    printf("    %-13s = %s\n", "score", F3(out.score));
    for (int i = 0; i < stats::attrs; ++i) {
        printf("    %-13s = %s\n", stats::names[i], F3(arr[i]));
    }
    printf("\n");
}


class TestGroup {
    std::vector<std::string> names;
    std::vector<stats> scores;

public:
    void combine(const TestGroup &other) {
        for (int i = 0; i < other.names.size(); ++i) {
            names.push_back(other.names[i]);
            scores.push_back(other.scores[i]);
        }
    }

    void add(const std::string &name, const char* kb) {
        names.push_back(name);
        scores.push_back(test(kb));
    }

    void print() const {
        const int n = scores.size();
        std::vector seen(n, false);

        for (int i = 0; i < n; ++i) {
            int best = -1;
            score_t cmp = 0;
            for (int j = 0; j < n; ++j) {
                if (seen[j]) continue;

                if (best == -1 || CMP(scores[j].score < cmp)) {
                    best = j;
                    cmp = scores[j].score;
                }
            }
            show(names[best], scores[best]);
            seen[best] = true;
        }
    }
};
#define TEST_(_name,s) { \
    constexpr char arr_[] = {s}; \
    t.add(#_name, arr_); \
}
#define TEST_T(name, s) TEST_(test{##name##},s)
#define TEST_N(name, s) TEST_(name,s)
TestGroup testNew_() {
    TestGroup t;

    TEST_T(dist-w&p,
        "1234567890"
        "j.glykcub;"
        "ietrhdnosa"
        "/xwmzpf,vq"
    );
    TEST_T(dist-full,
        "1234567890"
        "jbglykcu,;"
        "istrhdnoea"
        "/vwmzpf.xq"
    );
    TEST_T(okp1,
        "1234567890"
        "/oupqxlbfw"
        "eainkmhtsc"
        ";.,yzjrdvg"
    );
    TEST_T(okp2,
        "1234567890"
        ".uofqjmdbg"
        "ieanylrhts"
        ",;/pzwcvkx"
    );
    return t;
}

void test(const std::string &name, const char* keyboard) {
    TestGroup t;
    t.add(name, keyboard);
    t.print();
}

TestGroup testOther_() {
    TestGroup t;
    TEST_(qwerty,
        "1234567890"
        "qwertyuiop"
        "asdfghjkl;"
        "zxcvbnm,./"
    )
    TEST_(alphabet,
        "1234567890"
        "abcdefghij"
        "klmnopqrs;"
        "tuvwxyz,./"
    )
    TEST_(dvorak,
        "1234567890"
        "/,.pyfgcrl"
        "aoeuidhtns"
        ";qjkxbmwvz"
    )
    TEST_(colemak,
        "1234567890"
        "qwfpgjluy;"
        "arstdhneio"
        "zxcvbkm,./"
    )
    TEST_(carpalx,
        "1234567890"
        "qgmlwbyuv;"
        "dstnriaeoh"
        "zxcfjkp,./"
    )
    TEST_(arensito,
        "1234567890"
        "ql,p/;fudk"
        "arenbgsito"
        "zw.hjvcymx"
    )
    TEST_(asset,
        "1234567890"
        "qwjfgypul;"
        "asetdhnior"
        "zxcvbkm,./"
    )
    TEST_(capewell,
        "1234567890"
        ".ywdfjpluq"
        "aersgbtnio"
        "xzcv;kwh,/"
    )
    // TEST_(dickens,
    // "1 2 3 4 5 6 7 8 9 0 q"
    // "y p o u - v d l c w x"
    // "i n e a , m h t s r \""
    // "( ) ; . _ k f g b '"
    // "/ =             z j
    // )
    return t;
}
void testAll() {
    TestGroup t = testNew_();
    t.combine(testOther_());
    t.print();
}

void testUser() {
    char valid[2 * KEYS + 2] = {};
    memcpy(valid,            KEYS_LOWER, sizeof(KEYS_LOWER));
    memcpy(valid + KEYS + 1, KEYS_UPPER, sizeof(KEYS_UPPER));
    valid[KEYS] = '\n';
    printf("To test a keyboard, enter a string of text with exactly %d letters included in"
           "the following string:\n%s\nTo exit the testing loop, enter \"stop\"."
           , KEYS, valid);
    while (true) {
        char keyboard[KEYS];
        int seen = 0;

        printf("Enter a keyboard or \"stop\"");
        while (seen != KEYS) {
            char buffer[1024];
            std::cin.getline(buffer, 1024);
            if (buffer == "stop") {
                printf("Exiting.\n");
                goto exit;
            }
            for (int i = 0; buffer[i] != '\0'; ++i) {
                if (const int pos = letterUtils.positionOf(buffer[i]); pos != -1) {
                    if (seen < KEYS) keyboard[pos] = seen++;
                    else seen++;
                }
            }
            if (seen > KEYS) {
                printf("Too many keys found. Expected %d, found at least %d.\n", KEYS, seen);
            }
        }

        printf("Your keyboard:\n");
        printArrQ(keyboard);
        test("result", keyboard);
    }
    exit:
}