#include "hip/hip_runtime.h"
#include <lib/def.cuh>
#include <lib/text.cuh>
#include <common.cuh>
#include <metric.cuh>
#include <vector>


__global__ void setTest(keyboard* key, char* t) {
    key->arr = t;
    key->stats = {};
}

__global__ void getTest(const keyboard* k, stats* out) {
    *out = k->stats;
}

stats test(const char* kb) {
    char t[KEYS];
    mapKeys(kb, t);

    return score(t);
}

void show(const std::string &string, const stats &out) {
    const text_t* arr = (text_t*) ((char*) &out + sizeof(out.score));

    printf("%s:\n", string.c_str());
    printf("    %-13s = %s\n", "score", F3(out.score));
    for (int i = 0; i < stats::attrs; ++i) {
        printf("    %-13s = %s\n", stats::names[i], F3(arr[i]));
    }
    printf("\n");
}


class TestGroup {
    std::vector<const char*> names;
    std::vector<stats> scores;

public:
    void add(const char* name, const char* kb) {
        names.push_back(name);
        scores.push_back(test(kb));
    }

    void print() const {
        const int n = scores.size();
        std::vector seen(n, false);

        for (int i = 0; i < n; ++i) {
            int best = -1;
            score_t cmp = 0;
            for (int j = 0; j < n; ++j) {
                if (seen[j]) continue;

                if (best == -1 || CMP(scores[j].score < cmp)) {
                    best = j;
                    cmp = scores[j].score;
                }
            }
            show(names[best], scores[best]);
            seen[best] = true;
        }
    }
};
#define TEST_(_name,s) { \
    constexpr char arr_[] = {s}; \
    t.add(#_name, arr_); \
}
#define TEST_T(name, s) TEST_(test{##name##},s)
#define TEST_N(name, s) TEST_(name,s)
void testNew() {
    TestGroup t;

    TEST_T(dist-w&p,
        "1234567890"
        "j.glykcub;"
        "ietrhdnosa"
        "/xwmzpf,vq"
    );
    TEST_T(dist-full,
        "1234567890"
        "jbglykcu,;"
        "istrhdnoea"
        "/vwmzpf.xq"
    );
    TEST_T(okp1,
        "1234567890"
        "/oupqxlbfw"
        "eainkmhtsc"
        ";.,yzjrdvg"
    );
    TEST_T(okp2,
        "1234567890"
        ".uofqjmdbg"
        "ieanylrhts"
        ",;/pzwcvkx"
    );
    t.print();
}

void testOther() {
    TestGroup t;
    TEST_(qwerty,
        "1234567890"
        "qwertyuiop"
        "asdfghjkl;"
        "zxcvbnm,./"
    )
    TEST_(alphabet,
        "1234567890"
        "abcdefghij"
        "klmnopqrs;"
        "tuvwxyz,./"
    )
    TEST_(dvorak,
        "1234567890"
        "/,.pyfgcrl"
        "aoeuidhtns"
        ";qjkxbmwvz"
    )
    TEST_(colemak,
        "1234567890"
        "qwfpgjluy;"
        "arstdhneio"
        "zxcvbkm,./"
    )
    TEST_(carpalx,
        "1234567890"
        "qgmlwbyuv;"
        "dstnriaeoh"
        "zxcfjkp,./"
    )
    TEST_(arensito,
        "1234567890"
        "ql,p/;fudk"
        "arenbgsito"
        "zw.hjvcymx"
    )
    TEST_(asset,
        "1234567890"
        "qwjfgypul;"
        "asetdhnior"
        "zxcvbkm,./"
    )
    TEST_(capewell,
        "1234567890"
        ".ywdfjpluq"
        "aersgbtnio"
        "xzcv;kwh,/"
    )
    TEST_(dickens,
        "1234567890"
        "qwfpgjluy;"
        "arstdhneio"
        "zxcvbkm,./"
    )
    t.print();
}
