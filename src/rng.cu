#include "hip/hip_runtime.h"
#include "rng.cuh"

__managed__ __device__ state SEED;

#ifdef RNG64
constexpr __constant__ state RS[] {
    0x6f17b935f6e98b7fULL, 0x6a488b3741fbb86bULL, 0x4b0731b10ff131ebULL, 0x527316c55f3b941fULL, 0x570e5c965a13afebULL,
    0x4685c99fc65418b3ULL, 0x5539e1c873b0d283ULL, 0x787e4a132e5ce993ULL, 0x5711a9f8d80e9c87ULL, 0x4a18f5fc28a25091ULL,
};
constexpr __constant__ state RK[] {
    0x5b737c9ec727d301ULL, 0x7fe12909b1f3d897ULL, 0x579ee157f16ee655ULL, 0x476882d452065bcbULL, 0x60831701bde193c9ULL,
    0x6a8f1ed0f79bd6e9ULL, 0x676e537a3bb81939ULL, 0x4fa85f2022ec6a13ULL, 0x524a0812d4f5b4cbULL, 0x655fe226ea0e503fULL,
    0x724b21b6c4b76d07ULL, 0x5e7ef0e675a3a8d3ULL, 0x5b0d5536e8e823e1ULL, 0x4db34e8c878ac7fdULL, 0x4869980fd0eccaa7ULL,
    0x70975fc99ed257b9ULL, 0x5719c56fd5db0403ULL, 0x48f2221fdee011cfULL, 0x48fe5fcbd32a5ee3ULL, 0x448318ba956bb0cdULL,
    0x620104f07717ca9bULL, 0x5cb995e193184985ULL, 0x50989e4c8a7197c7ULL, 0x4d89e496826e3237ULL, 0x656a77a176a220b1ULL,
    0x789b1d131ccbd0fbULL, 0x689e5076328c5543ULL, 0x66e553d500b2682fULL, 0x5028e21c00f54bd9ULL, 0x4d6626bb2c0b81b9ULL,
};

__device__ state next(state &x) {
    x ^= x >> 12;
    x ^= x << 25;
    x ^= x >> 27;
    return x * 0x2545F4914F6CDD1DULL;
}
#else
constexpr __constant__ state RS[] {
    0x4d06ca3d, 0x63648235, 0x706a1dbd, 0x5f67fca9, 0x4b6ef69b,
    0x5bfbc19f, 0x55dd80ff, 0x461c76bf, 0x7b30f553, 0x6a4d6485,
};
constexpr __constant__ state RK[] {
    0x51f16c93, 0x4c6757b1, 0x4bfd1e15, 0x5494f35b, 0x61b5f267,
    0x4bc3373f, 0x4a928961, 0x42e03105, 0x5da28279, 0x7cb2e205,
    0x76367053, 0x422b5863, 0x5189927b, 0x716a5189, 0x4875ca91,
    0x6e517733, 0x515dd9a5, 0x4c30a4c7, 0x74309497, 0x7b09be39,
    0x5f9e4dad, 0x7e922e6d, 0x7acfb43b, 0x58af1107, 0x48ea7277,
    0x67d1bf41, 0x6875b0ed, 0x4540ff91, 0x539c0825, 0x7e3a641b,
};

__device__ state next(state &x) {
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;
    return x;
}
#endif
__device__ float nextf(state &x) {
    return next(x) / (float) (state) -1;
}

__device__ state hash(const char* arr, const state i) {
    state s = RS[0] * (SEED * i + RS[1]);
    #pragma unroll
    for (int j = 0; j < KEYS; ++j) {
        s ^= (2 + arr[j]) * RK[j];
    }
    return s * (SEED + RS[9]);
}

__device__ state hash(const state x, const state y) {
    return x * RS[6] ^ y * RS[7];
}

__device__ state hash(const state x, const state y, const state z) {
    return x * RS[6] ^ y * RS[7] ^ z * RS[8];
}

__global__ void updateStateLocal(const char* kb, const state offset) {
    state s = hash(kb, offset) * RS[2];
    state out = RS[3] * SEED;
    for (int i = 0; i < 3; i++) out = out * RS[4] ^ next(s) * RS[5];
#ifdef RNG64
    SEED = out;
#else
    SEED = out >> 32 ^ out;
#endif
}

__host__ void updateState(const char* kb, const state offset) {
    updateStateLocal<<<1, 1>>>(kb, offset);
    hipDeviceSynchronize();
}
